// vectorizing addition
// indexing threads

#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>

using std::begin;
using std::copy;
using std::cout;
using std::end;
using std::endl;
using std::generate;
using std::vector;

// CUDA kernel for vector addition
__global__ void vectorAdd(int* a, int* b, int* c, int N) {
    // calculate global ID thread
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // bounds check
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    // array size of 2^16
    constexpr int N = 1 << 16;
    size_t bytes = sizeof(int) * N;

    // vectors for holding host-side data
    vector<int> a(N);
    vector<int> b(N);
    vector<int> c(N);

    // initialize numbers in each array
    generate(begin(a), end(a), []() {return rand() % 100; });
    generate(begin(b), end(b), []() {return rand() % 100; });

    // allocate memory on device
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    //copy data from the host to the device (CPU -> GPU)
    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);
    // dont need c vector, will be created on GPU

    // threads per CTA (1024 threads per CTA)
    int NUM_THREADS = 1 << 10;

    // CTAs per grid
    // we need to launch at least as many threads as we have elements
    // This equation pads and extra CTA to the grid if N cannot evenly be divided
    // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

    // Launch the kerned on the GPU
    // kernel calls are asynchronous (The CPU Program cintinues execution after call
    // but not necessarily before the kernel finishes)
    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

    // Copy Sum vector from device to host
    // cudaMemcpy is a synchronous operation, and waits for the prior kernel
    // launch to complete (both go to the default stream in this case).
    // therefore, this cudaMemcpy acts as both a memcpy and synchronization barrier
    hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    // Check result for errors
//    verify)results(a,b,c)

}